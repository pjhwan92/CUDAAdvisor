#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>


//extern __device__ int testxyz[1000];
//int localtrace[10000];
//__device__ float* tracehandle;

__device__ float foo_CC(float a)
{
	return a*0.9;
}

__device__ int foo_DD(float a)
{
	if (threadIdx.x < 2 || threadIdx.y > 2)
		return (int) a;
	else
		return a+2;
}

__device__ float foo_BB(float a)
{
	if (threadIdx.x > 3 || threadIdx.y > 11)
		return a + foo_CC(a);
	else 
		return a + (float)foo_DD(a) /2;
}

__device__ float foo_AA( float a, float b)
{
	if (threadIdx.x < 8  || threadIdx.y > 4)
		return a*3.1415+1;
	else
		return (b*a)*0.5 + foo_BB(b);
}


__global__ void axpy_kernel2(float a, float* x, float* y)
{
	//tracehandle = newbu;
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	int index = threadId;

	float aa = y[index] + x[index] + 1.1;
	float b = 0.5*y[index] + 0.25* x[index] + 1.0;
	y[index] += ( x[index]*1.67 +  foo_AA(aa, b) );
	//	y[index] += ( x[index]*1.67 +  aa + b );
}

__global__ void axpy_kernel1(float a, float* x, float* y) 
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	int index = threadId;

	y[index] = x[index]*0.3;

	if (index>2)
		y[index] += 99;
	else
		y[index] += 999 + foo_CC(a);

}

int main(int argc, char* argv[]) 
{
	//const int kDataLen2 = 128;
	float a = 2.0f;
	//int blocks2 = 600; 
	hipSetDevice(0);

	if (argc != 5)
	{
		printf("usage: ./axpy [blocks_x] [blocks_y] [threads_x] [threads_y]\n");
		exit(1);
	}

	int blocksx = atoi(argv[1]) ;
	int blocksy = atoi(argv[2]) ;
	int kDataLenx = atoi(argv[3]);
	int kDataLeny = atoi(argv[4]);

	int sizen = blocksx *blocksy *kDataLenx *kDataLeny;
	//	cudaThreadSetLimit(cudaLimitMallocHeapSize, 1024*1024); //sderek
	hipDeviceSetLimit(hipLimitMallocHeapSize, 1024*1024*500); //sderek

	//	tracetest = (int*)malloc( 1234);

	//	float host_y[blocks*kDataLen];
	//	float host_y[blocks*kDataLen];
	float* host_x = (float*) malloc( sizen* sizeof(float));
	float* host_y = (float*) malloc( sizen* sizeof(float));
	void* host_newbu = (void*) malloc( 1000 );

	int ii;
	for( ii=0; ii<sizen; ii++)
		host_x[ii] = ii%8;
	for( ii=0; ii<sizen; ii++)
		host_y[ii] = ii%5;
	/*	int x[5];
			x[0] = 13;
			printf("%p\n",x);
			printf("%p\n",&x);
			printf("%d\n",*x);
			printf("%d\n",*(x+1));
	 */

	// Copy input data to device.
	float* device_x;
	float* device_y;
	//	void* newbu;

	//	printf(" %p\n", device_x);
	hipMalloc((void**)&device_x, sizen * sizeof(float));
	//	printf(" %p\n", device_x);
	//	printf(" %p\n", device_y);
	hipMalloc((void**)&device_y, sizen * sizeof(float) + 18);
	//	printf(" %p\n", device_y);

	//	printf(" %p\n", newbu);
	//	cudaMalloc(&newbu, 1000);
	//	printf(" %p\n", newbu);

	/*	std::cout << &(device_x) << "\n";
			std::cout << &(device_y) << "\n";
			std::cout << &(*device_x) << "\n";
			std::cout << &(*device_y) << "\n";
			std::cout << (device_x) << "\n";
			std::cout << (device_y) << "\n";
	 */
	hipMemcpy(device_x, host_x, sizen * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_y, host_y, sizen * sizeof(float), hipMemcpyHostToDevice);

	dim3 CTAs(blocksx, blocksy);
	dim3 Threads(kDataLenx, kDataLeny);
	std::cout << "launching kernel...\n";
	axpy_kernel1<<<CTAs, Threads>>>(a, device_x, device_y);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf ("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}

	axpy_kernel2<<<CTAs, Threads>>>(a, device_x, device_y);
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf ("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	hipMemcpy(host_y, device_y, sizen* sizeof(float), hipMemcpyDeviceToHost);

	//	cudaMemcpy(host_newbu, newbu, 1000, cudaMemcpyDeviceToHost);

	free(host_newbu);
	//	cudaFree(newbu);

	int verify = 0;
	for (int ii = 0; ii < 8; ii++)
		std::cout << "y[" << ii << "] = " << host_y[ii] << "\n";

	for (int ii = 0; ii < sizen; ii++) {
		if( host_y[ii] == ii%5)
			verify ++;
		//    		std::cout << "y[" << i << "] = " << host_y[i] << "\n";
	}
	std::cout << "\n\n[TOOL verify] There are a total of\t" << verify << " incorrect numbers." << std::endl;
	if (verify==0)
		std::cout << "[TOOL verify] passed!" << std::endl << std::endl;



	//	for (int i = 0; i < 20; ++i) {
	//                std::cout << "newtrace [" << i << "] = " << host_newbu[i] << "\n";
	//    std::cout << & (host_y[i] )<< "\n";
	//        }

	/*        cudaMemcpyFromSymbol(localtrace, testxyz, 40*sizeof(int), 0, cudaMemcpyDeviceToHost);
						for (int i = 0; i < 20; ++i)
						printf("%d\t", localtrace[i] );
						std::cout << std::endl;

						cudaMemcpyFromSymbol(localtrace+8, testxyz, 40*sizeof(int), 0, cudaMemcpyDeviceToHost);
						for (int i = 0; i < 20; ++i)
						printf("%d\t", localtrace[i] );
						std::cout << std::endl;
	 */
	//	int* show_h;
	//	cudaMemcpyFromSymbol(show_h, show, sizeof(int), 0, cudaMemcpyDeviceToHost);
	//	msg = cudaGetSymbolAddress((void **)&d_G, test);
	//		printf("the address is %p\n", d_G);
	//	if (msg == cudaSuccess)
	//	{
	//	int tmp[4];
	//	printf("before %d %d %d %d@ %p\n", *tmp,  *(tmp+1), *(tmp+2), *(tmp+3), tmp);
	//	cudaMemcpyFromSymbol(tracetest, test1, 4*sizeof(int), 0, cudaMemcpyDeviceToHost);
	//	cudaMemcpyFromSymbol(tmp, test2, 4*sizeof(int), 0, cudaMemcpyDeviceToHost);
	//	printf("copy %d %d %d %d@ %p\n",  *tmp, *(tmp+1), *(tmp+2), *(tmp+3), tmp);
	//	cudaMemcpyFromSymbol(tmp, test2, 4*sizeof(int), 0, cudaMemcpyDeviceToHost);
	//	printf("after %d %d %d %d@ %p\n",  tmp[0], tmp[1], tmp[2], tmp[3], tmp);
	//	}
	//else
	//	std::cout << cudaGetErrorString(msg)  <<  "\n\n";

	hipFree(device_x);
	hipFree(device_y);
	hipDeviceReset();
	return 0;
}
